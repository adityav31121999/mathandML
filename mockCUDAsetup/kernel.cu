#include "hip/hip_runtime.h"

#include "header.hpp"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

// Kernel 1: Addition
__global__ void add_vectors(const float *a,
                            const float *b,
                            float *c,
                            const unsigned int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] + b[i];
    }
}

// Kernel 2: Subtraction
__global__ void subtract_vectors(const float *a,
                                 const float *b,
                                 float *c,
                                 const unsigned int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        c[i] = a[i] - b[i];
    }
}
