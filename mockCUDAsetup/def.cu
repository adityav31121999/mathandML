#include "hip/hip_runtime.h"

#include "header.hpp"
#include <fstream>
#include <stdexcept> // Ensure stdexcept is included for runtime_error
#include <cmath>     // For ceil

/**
 * @brief Adds vectors using a pre-compiled CUDA kernel.
 * @param a vector input for addition
 * @param b vector input for addition
 * @param res result to store vector addition
 * @note Assumes the 'add_vectors' CUDA kernel is compiled and linked.
 */
void addVectorsCUDA(std::vector<float>& a, std::vector<float>& b, std::vector<float>& res) {
    if (a.size() != b.size() || a.size() != res.size()) {
        throw std::runtime_error("Vector sizes do not match.");
    }

    unsigned int vector_size = static_cast<unsigned int>(a.size()); // Kernel expects unsigned int
    size_t vector_bytes = sizeof(float) * vector_size;

    // Initialize CUDA (selects device)
    HIPContext cudaCtx; // Uses device 0 by default

    // Allocate memory on the CUDA device
    float *d_a = nullptr, *d_b = nullptr, *d_res = nullptr;
    CUDA_CHECK(hipMalloc(&d_a, vector_bytes));
    CUDA_CHECK(hipMalloc(&d_b, vector_bytes));
    CUDA_CHECK(hipMalloc(&d_res, vector_bytes));
    std::cout << "CUDA Memory Allocated." << std::endl;

    // Copy data from host (std::vector) to device (CUDA pointers)
    CUDA_CHECK(hipMemcpy(d_a, a.data(), vector_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, b.data(), vector_bytes, hipMemcpyHostToDevice));
    std::cout << "Data Copied Host -> Device." << std::endl;

    // Define kernel launch configuration (Grid and Block dimensions)
    int threadsPerBlock = 256;
    // Calculate grid size to cover all elements
    int blocksPerGrid = (vector_size + threadsPerBlock - 1) / threadsPerBlock;
    dim3 gridDim(blocksPerGrid);
    dim3 blockDim(threadsPerBlock);

    // Launch the kernel
    add_vectors<<<gridDim, blockDim, 0, cudaCtx.stream>>>(d_a, d_b, d_res, vector_size);
    CUDA_CHECK(hipGetLastError()); // Check for kernel launch errors
    std::cout << "CUDA Kernel Launched." << std::endl;

    // Copy results from device back to host
    CUDA_CHECK(hipMemcpy(res.data(), d_res, vector_bytes, hipMemcpyDeviceToHost));
    std::cout << "Data Copied Device -> Host." << std::endl;

    // Synchronize to ensure kernel and copies are finished (optional but good practice)
    // CUDA_CHECK(hipStreamSynchronize(cudaCtx.stream)); // Sync specific stream
    CUDA_CHECK(hipDeviceSynchronize()); // Sync entire device (simpler for this case)

    // Free device memory
    CUDA_CHECK(hipFree(d_a));
    CUDA_CHECK(hipFree(d_b));
    CUDA_CHECK(hipFree(d_res));
    std::cout << "CUDA Memory Freed." << std::endl;
}


/**
 * @brief Adds vectors using a pre-compiled CUDA kernel.
 * @param a vector input for addition
 * @param b vector input for addition
 * @param res result to store vector addition
 * @note Assumes the 'add_vectors' CUDA kernel is compiled and linked.
 */
void subtractVectorsCUDA(std::vector<float>& a, std::vector<float>& b, std::vector<float>& res) {
    if (a.size() != b.size() || a.size() != res.size()) {
        throw std::runtime_error("Vector sizes do not match.");
    }

    unsigned int vector_size = static_cast<unsigned int>(a.size()); // Kernel expects unsigned int
    size_t vector_bytes = sizeof(float) * vector_size;

    // Initialize CUDA (selects device)
    HIPContext cudaCtx; // Uses device 0 by default

    // Allocate memory on the CUDA device
    float *d_a = nullptr, *d_b = nullptr, *d_res = nullptr;
    CUDA_CHECK(hipMalloc(&d_a, vector_bytes));
    CUDA_CHECK(hipMalloc(&d_b, vector_bytes));
    CUDA_CHECK(hipMalloc(&d_res, vector_bytes));
    std::cout << "CUDA Memory Allocated." << std::endl;

    // Copy data from host (std::vector) to device (CUDA pointers)
    CUDA_CHECK(hipMemcpy(d_a, a.data(), vector_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_b, b.data(), vector_bytes, hipMemcpyHostToDevice));
    std::cout << "Data Copied Host -> Device." << std::endl;

    // Define kernel launch configuration (Grid and Block dimensions)
    int threadsPerBlock = 256;
    // Calculate grid size to cover all elements
    int blocksPerGrid = (vector_size + threadsPerBlock - 1) / threadsPerBlock;
    dim3 gridDim(blocksPerGrid);
    dim3 blockDim(threadsPerBlock);

    // Launch the kernel
    subtract_vectors<<<gridDim, blockDim, 0, cudaCtx.stream>>>(d_a, d_b, d_res, vector_size);
    CUDA_CHECK(hipGetLastError()); // Check for kernel launch errors
    std::cout << "CUDA Kernel Launched." << std::endl;

    // Copy results from device back to host
    CUDA_CHECK(hipMemcpy(res.data(), d_res, vector_bytes, hipMemcpyDeviceToHost));
    std::cout << "Data Copied Device -> Host." << std::endl;

    // Synchronize to ensure kernel and copies are finished (optional but good practice)
    // CUDA_CHECK(hipStreamSynchronize(cudaCtx.stream)); // Sync specific stream
    CUDA_CHECK(hipDeviceSynchronize()); // Sync entire device (simpler for this case)

    // Free device memory
    CUDA_CHECK(hipFree(d_a));
    CUDA_CHECK(hipFree(d_b));
    CUDA_CHECK(hipFree(d_res));
    std::cout << "CUDA Memory Freed." << std::endl;
}
